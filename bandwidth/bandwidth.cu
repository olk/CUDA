#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <stdexcept>

#include <hip/hip_runtime.h>

#include <numa.h>

float cuda_malloc_test( int size, bool up, int numa_node) {
    hipEvent_t start, stop;
    hipEventCreate( & start);
    hipEventCreate( & stop);

    int * a = static_cast< int * >( ::numa_alloc_onnode( size * sizeof( int), numa_node) );
    int * dev_a;
    hipMalloc( & dev_a, size * sizeof( * dev_a ) );

    hipEventRecord( start, 0);
    for ( int i = 0; i < 100; ++i) {
        if ( up) {
            hipMemcpy( dev_a, a, size * sizeof( * dev_a), hipMemcpyHostToDevice);
        } else {
            hipMemcpy( a, dev_a, size * sizeof( * dev_a), hipMemcpyDeviceToHost);
        }
    }
    hipEventRecord( stop, 0);
    hipEventSynchronize( stop);
    float elapsedTime;
    hipEventElapsedTime( & elapsedTime, start, stop);

    numa_free( a, size * sizeof( int) );
    hipFree( dev_a);
    hipEventDestroy( start);
    hipEventDestroy( stop);

    return elapsedTime;
}

int main( int argc, char * argv[]) {
    int numa_node = std::atoi( argv[1]);
    ::numa_run_on_node( numa_node);
    std::printf("NUMA node: %d\n", numa_node);
    for ( int dev = 0; dev < 2; ++dev) {
        hipSetDevice( dev);

        constexpr int size = 32 * 1024 * 1024 * sizeof( int);
        constexpr float gb = static_cast< float >( 100) * size * sizeof( int)/(1024 * 1024 * 1024);

        int curr_dev = -1;
        hipGetDevice( & curr_dev);
        std::printf("GPU: %d\n", curr_dev);

        float elapsedTime = cuda_malloc_test( size, true, numa_node);
        std::printf("host-to-device: %3.1f GB/s\n", gb / (elapsedTime/1000) );

        elapsedTime = cuda_malloc_test( size, false, numa_node);
        std::printf("device-to-host: %3.1f GB/s\n", gb / (elapsedTime/1000) );
    }

    return EXIT_SUCCESS;
}
